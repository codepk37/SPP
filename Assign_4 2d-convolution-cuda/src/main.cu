
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <hip/hip_runtime.h>

namespace solution {
    #define CUDA_ERROR_CHECK(ans) { cudaAssert((ans), __FILE__, __LINE__); } 

    inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
        if (code != hipSuccess) {
            fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }

    __constant__ float kernel[3][3] = {
        { 0.0625f, 0.125f, 0.0625f },
        { 0.125f, 0.25f, 0.125f },
        { 0.0625f, 0.125f, 0.0625f }
    }; // Define kernel as constant memory

    __global__ void convolution2D(const float* img, float* output, int num_rows, int num_cols) {
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;

        // Calculate global indices
        int row = by * blockDim.y + ty;
        int col = bx * blockDim.x + tx;

        // Perform convolution
        if (row < num_rows && col < num_cols) {
            float sum = 0.0f;
            for (int i = -1; i <= 1; ++i) {
                for (int j = -1; j <= 1; ++j) {
                    int row_idx = row + i;
                    int col_idx = col + j;
                    if (row_idx >= 0 && row_idx < num_rows && col_idx >= 0 && col_idx < num_cols) {
                        sum += kernel[i + 1][j + 1] * img[row_idx * num_cols + col_idx];
                    }
                }
            }
            output[row * num_cols + col] = sum;
        }
    }

    std::string compute(const std::string& bitmap_path, const float kernel[3][3], const std::int32_t num_rows, const std::int32_t num_cols) {
        std::string sol_path = std::filesystem::temp_directory_path() / "student_sol.bmp";
        std::ofstream sol_fs(sol_path, std::ios::binary);
        std::ifstream bitmap_fs(bitmap_path, std::ios::binary);
        const auto img = std::make_unique<float[]>(num_rows * num_cols);
        bitmap_fs.read(reinterpret_cast<char*>(img.get()), sizeof(float) * num_rows * num_cols);
        bitmap_fs.close();

        // Allocate memory on the GPU
        float* d_img;
        float* d_output;
        CUDA_ERROR_CHECK(hipMalloc(&d_img, sizeof(float) * num_rows * num_cols));
        CUDA_ERROR_CHECK(hipMalloc(&d_output, sizeof(float) * num_rows * num_cols));

        // Transfer data to GPU
        CUDA_ERROR_CHECK(hipMemcpy(d_img, img.get(), sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

        // Define block and grid dimensions
        dim3 blockDim(32, 32); // Adjust block size as needed
        dim3 gridDim((num_cols + blockDim.x - 1) / blockDim.x, (num_rows + blockDim.y - 1) / blockDim.y);

        // Call CUDA kernel
        convolution2D<<<gridDim, blockDim>>>(d_img, d_output, num_rows, num_cols);
        CUDA_ERROR_CHECK(hipGetLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());

        // Transfer result back to CPU
        float* output = new float[num_rows * num_cols];
        CUDA_ERROR_CHECK(hipMemcpy(output, d_output, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

        // Write output to file
        sol_fs.write(reinterpret_cast<char*>(output), sizeof(float) * num_rows * num_cols);

        // Cleanup
        CUDA_ERROR_CHECK(hipFree(d_img));
        CUDA_ERROR_CHECK(hipFree(d_output));
        delete[] output;

        sol_fs.close();
        return sol_path;
    }
}
